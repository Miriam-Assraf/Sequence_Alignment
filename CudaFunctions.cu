#include "hip/hip_runtime.h"
/* Miriam Assraf */

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

__global__ void calcScoresKernel(double* dev_results, double* dev_scores, int num_rows, int num_cols);
__host__ void checkErrors(hipError_t err, const char* error_msg);

int calcScoreWithCuda(double* similarities, double* scores, int num_rows, int num_cols)
{
	hipError_t err = hipSuccess;
	double* dev_similarities;
	double* dev_scores;

	// Allocate memory on GPU 
	err = hipMalloc((void**)&dev_similarities, num_rows * num_cols * sizeof(double));
	checkErrors(err, "Failed to allocate similarities to device memory - %s\n");

	err = hipMalloc((void**)&dev_scores, num_rows * sizeof(double));
	checkErrors(err, "Failed to allocate scores to device memory - %s\n");
	// Initialize device scores with zeros
	err = hipMemset(dev_scores, 0, num_rows * sizeof(double));
	checkErrors(err, "Failed to initiate device scores - %s\n");

	// Copy similarities from host to the GPU memory
	err = hipMemcpy(dev_similarities, similarities, num_rows * num_cols * sizeof(double), hipMemcpyHostToDevice);
	checkErrors(err, "Failed to copy similarities from host to device - %s\n");

	// Launch the Kernel
	int threadsPerBlock = (int)ceil(sqrt(num_rows));
	if (threadsPerBlock % 32 != 0)
	{
		threadsPerBlock = threadsPerBlock + 32 - threadsPerBlock % 32;	// Make sure block size is multiple of 32
	}
	int blocksPerGrid = (num_rows + threadsPerBlock - 1) / threadsPerBlock;	// We get total number of threads as number of rows (or a bit more so it is a multiple of 32)
	calcScoresKernel << <blocksPerGrid, threadsPerBlock >> > (dev_similarities, dev_scores, num_rows, num_cols);
	err = hipGetLastError();
	checkErrors(err, "Failed to launch comparison kernel -  %s\n");
	
	// Copy the  scores from GPU to the host memory
	err = hipMemcpy(scores, dev_scores, num_rows * sizeof(double), hipMemcpyDeviceToHost);
	checkErrors(err, "Failed to copy calculated scores from device to host -%s\n");
	
	// Free allocated memory on GPU
	err = hipFree(dev_similarities);
	checkErrors(err, "Failed to free device similarities - %s\n");

	err = hipFree(dev_scores);
	checkErrors(err, "Failed to free device scores - %s\n");

	return 0;
}

__global__ void calcScoresKernel(double* dev_similarities, double* dev_scores, int num_rows, int num_cols)
{
	// Each thread calculates score for one row of similarities (one mutant similarity)
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	for (int i = 0; i < num_cols; i++) {
		dev_scores[row] += dev_similarities[num_cols * row + i];
	}
}

__host__ void checkErrors(hipError_t err, const char* error_msg)
{
	// If didn't return hipSuccess print error
	if (err != hipSuccess) {
		fprintf(stderr, error_msg, hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}
